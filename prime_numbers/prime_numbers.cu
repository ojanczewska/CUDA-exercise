#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <omp.h>
#include <chrono>

using namespace std;


bool prime_CPU(unsigned long long int a);
bool prime_GPU(unsigned long long int a, bool gpu_prime);

__global__ void kernel(unsigned long long int* d_a, bool* d_c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	i = 2 * i + 1;
	if (*d_a % i == 0) {
		*d_c = false;
	}
}
__global__ void  dummy() {

}

bool prime_OMP(unsigned long long int a) {
	bool prime = true;
	long long int pierwiastek = sqrt(a) + 2;

#pragma omp parallel for
	for (long long i = 3; i < pierwiastek; i = i + 2) {
		if (a % i == 0) {
			prime = false;
			break;
		}
	}



	return prime;


}


int main()
{
	//unsigned long long int a;
	//printf("Podaj liczbe do sprawdzenia:");
	//scanf("%llu", &a);
	bool gpu_prime = true;
	bool gpu_prime_wsp = true;
	dummy << <1, 1 >> > ();


	unsigned long long int liczby_testowe[6] = { 524287 ,2147483647  ,274876858369  ,1125897758834689 ,2305843009213693951 ,4611686014132420609 };


	for (int i = 0; i < 6; i++)
	{
		unsigned long long int proba = liczby_testowe[i];
		//unsigned long long int proba = a;
		printf("\nLiczba testowa: %llu \n", proba);
		if (proba == 1 || proba % 2 == 0) {
			printf("Liczba nie jest pierwsza");
			return (0);
		}
		if (proba == 2) {
			printf("Liczba jest pierwsza");
			return (0);
		}
		printf("\n-------------CPU ----------------\n");

		auto CPUstart = chrono::steady_clock::now();
		bool prime_cmp = prime_CPU(proba);
		auto CPUend = chrono::steady_clock::now();
		chrono::duration<double> elapsedCPU = CPUend - CPUstart;

		if (prime_cmp == true) {
			printf("Podana liczba jest liczba pierwsza.");
		}
		else
		{
			printf("Podana liczba jest zlozona.");
		}
		printf("\nCode executed in %f ms.\n", elapsedCPU.count());

		printf("\n-------------CPU OpenMP----------------\n");

		auto CMPstart = chrono::steady_clock::now();
		bool prime_cpu = prime_OMP(proba);
		auto CMPend = chrono::steady_clock::now();
		chrono::duration<double> elapsedCMP = CMPend - CMPstart;

		if (prime_cpu == true) {
			printf("Podana liczba jest liczba pierwsza.");
		}
		else
		{
			printf("Podana liczba jest zlozona.");
		}
		printf("\nCode executed in %f ms.\n", elapsedCMP.count());



		printf("\n-------------GPU ----------------\n");

		auto GPUstart = chrono::steady_clock::now();
		bool gpu3 = prime_GPU(proba, gpu_prime);
		auto GPUend = chrono::steady_clock::now();
		chrono::duration<double> elapsedGPU = GPUend - GPUstart;

		if (gpu3 == true) {
			printf("Podana liczba jest liczba pierwsza.");
		}
		else
		{
			printf("Podana liczba jest zlozona.");
		}

		printf("\nCode executed in %f ms.\n\n", elapsedGPU.count());


	}
	return (0);
}
bool prime_CPU(unsigned long long int a) {
	bool prime_cpu = true;
	for (long int i = 3; i <= sqrt(a); i += 2) {
		if (a % i == 0) {
			prime_cpu = false;
			break;
		}
	}
	return prime_cpu;
}


bool prime_GPU(unsigned long long int a, bool gpu_prime) {
	unsigned long long int lim = sqrt(a);
	unsigned long long int* d_a = 0;
	bool* d_c;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&d_a, sizeof(unsigned long long int));
	cudaStatus = hipMalloc((void**)&d_c, sizeof(bool));

	cudaStatus = hipMemcpy(d_a, &a, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_c, &gpu_prime, sizeof(bool), hipMemcpyHostToDevice);
	long int x = (lim / 32 + 1) / 2;
	dim3 block(32);
	dim3 grid(x);
	kernel << <grid, block >> > (d_a, d_c);
	cudaStatus = hipMemcpy(&a, d_a, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(&gpu_prime, d_c, sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_c);
	return gpu_prime;
}



