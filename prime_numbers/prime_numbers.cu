#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>

bool prime_CPU(unsigned long long int a);
bool prime_GPU(unsigned long long int a, bool gpu_prime);
__global__ void kernel(unsigned long long int* d_a, bool* d_c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x + 3;
	if (*d_a % i == 0) {
		*d_c = false;
	}
}
int main()
{
	//unsigned long long int a;
	//printf("Podaj liczbe do sprawdzenia:");
	//scanf("%llu", &a);
	clock_t t;
	bool gpu_prime = true;
	unsigned long long int liczby_testowe[6] = { 524287 ,2147483647 ,2305843009213693951 ,274876858369 ,4611686014132420609 ,1125897758834689 };

	for (int i = 0; i < 6; i++)
	{
	    unsigned long long int proba = liczby_testowe[i];
		//unsigned long long int proba = a;
		printf("\nLiczba testowa: %llu \n", proba);
		if (proba == 1 || proba % 2 == 0) {
			printf("Liczba nie jest pierwsza");
			return (0);
		}
		if (proba == 2) {
			printf("Liczba jest pierwsza");
			return (0);
		}
		printf("\n-------------CPU----------------\n");
		t = clock();
		bool prime_cpu = prime_CPU(proba);
		t = clock() - t;
		double time = ((double)t) / CLOCKS_PER_SEC;
		if (prime_cpu == true) {
			printf("Podana liczba jest liczba pierwsza.");
		}
		else
		{
			printf("Podana liczba jest zlozona.");
		}
		printf("\nCzas : %f ", time);

		printf("\n-------------GPU----------------\n");
		t = clock();
		bool gpu = prime_GPU(proba, gpu_prime);
		t = clock() - t;
		time = ((double)t) / CLOCKS_PER_SEC;
		if (gpu == true) {
			printf("Podana liczba jest liczba pierwsza.");
		}
		else
		{
			printf("Podana liczba jest zlozona.");
		}
		printf("\nCzas : %f \n\n", time);

	}
	return (0);
}
bool prime_CPU(unsigned long long int a) {
	bool prime_cpu = true;
	for (long int i = 3; i <= sqrt(a); i += 2) {
		if (a % i == 0) {
			prime_cpu = false;
			break;
		}
	}
	return prime_cpu;
}
bool prime_GPU(unsigned long long int a, bool gpu_prime) {
	unsigned long long int lim = sqrt(a);
	unsigned long long int* d_a = 0;
	bool* d_c;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&d_a, sizeof(unsigned long long int));
	cudaStatus = hipMalloc((void**)&d_c, sizeof(bool));

	cudaStatus = hipMemcpy(d_a, &a, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_c, &gpu_prime, sizeof(bool), hipMemcpyHostToDevice);
	long int x = lim / 32 + 1;
	dim3 block(32);
	dim3 grid(x);
	kernel << <grid, block >> > (d_a, d_c);
	cudaStatus = hipMemcpy(&a, d_a, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(&gpu_prime, d_c, sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_c);
	return gpu_prime;
}



